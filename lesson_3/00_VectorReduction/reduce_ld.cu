#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"

using namespace timer;

// Macros
#define DIV(a, b)   (((a) + (b) - 1) / (b))

const int N  = 16777216;
#define BLOCK_SIZE 256

__global__ void ReduceKernelLowDivergence(int* VectorIN, int N) {
	__shared__ int SMem[1024];
	int GlobalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	SMem[threadIdx.x] = VectorIN[GlobalIndex];
	__syncthreads();

	for (int i = 1; i < blockDim.x; i *= 2) {
		int index = threadIdx.x * i * 2;
		if (index < blockDim.x)
			SMem[index] += SMem[index + i];
		__syncthreads();
	}
	if (threadIdx.x == 0)
		VectorOUT[blockIdx.x] = SMem[0];
}

int main() {
    
	// ------------------- INIT ------------------------------------------------

	// Random Engine Initialization
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::default_random_engine generator (seed);
	std::uniform_int_distribution<int> distribution(1, 100);

	Timer<HOST> host_TM;
	Timer<DEVICE> dev_TM;

	// ------------------ HOST INIT --------------------------------------------

	int* VectorIN = new int[N];
	for (int i = 0; i < N; ++i)
		VectorIN[i] = distribution(generator);

	// ------------------- CUDA INIT -------------------------------------------

	int* devVectorIN;
	SAFE_CALL( hipMalloc(&devVectorIN, N * sizeof(int)) );
	
	SAFE_CALL( hipMemcpy(devVectorIN, VectorIN, N * sizeof(int),
                 hipMemcpyHostToDevice) );
	
	int sum;
	float dev_time;

	// ------------------- CUDA COMPUTATION  ----------------------------------

	std::cout<<"Starting computation on DEVICE "<<std::endl;

	dev_TM.start();

	ReduceKernelLowDivergence<<<DIV(N, BLOCK_SIZE), BLOCK_SIZE>>>
		(devVectorIN, N);
	ReduceKernelLowDivergence<<<DIV(N, BLOCK_SIZE* BLOCK_SIZE), BLOCK_SIZE>>>
	 	(devVectorIN, DIV(N, BLOCK_SIZE));
	ReduceKernelLowDivergence<<<DIV(N, BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE), BLOCK_SIZE>>>
	 	(devVectorIN, DIV(N, BLOCK_SIZE * BLOCK_SIZE));

	dev_TM.stop();
	dev_time1 = dev_TM.duration();
	CHECK_CUDA_ERROR;

	SAFE_CALL( hipMemcpy(&sum, devVectorIN, sizeof(int),
	    	hipMemcpyDeviceToHost) );

	// ------------------- HOST ------------------------------------------------
	host_TM.start();

	int host_sum = std::accumulate(VectorIN, VectorIN + N, 0);

	host_TM.stop();

	std::cout << std::setprecision(3)
	      << "KernelTime Divergent: " << dev_time << std::endl
	      << "HostTime            : " << host_TM.duration() << std::endl
	      << std::endl;

	// ------------------------ VERIFY -----------------------------------------

	if (host_sum != sum) {
	std::cerr << std::endl
		  << "Error! Wrong result. Host value: " << host_sum
		  << " , Device value: " << sum
		  << std::endl << std::endl;
	hipDeviceReset();
	std::exit(EXIT_FAILURE);
	}

	//-------------------------- SPEEDUP ---------------------------------------

	float speedup = host_TM.duration() / dev_time;

	std::cout << "Correct result" << std::endl
	      << "Speedup achieved: " << std::setprecision(3)
	      << speedup << " x" << std::endl << std::endl;

	std::cout << host_TM.duration() << ";" << dev_TM.duration() << ";" << host_TM.duration() / 			dev_TM.duration() << std::endl;

	delete[] VectorIN;
	SAFE_CALL( hipFree(devVectorIN) );
	hipDeviceReset();
	}
